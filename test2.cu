
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

const int N = 1024;

// CUDA 커널 함수: 행렬 곱셈 함수
__global__ void matrixMultiplication(float* A, float* B, float* C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main() {
    float* A = new float[N * N];
    float* B = new float[N * N];
    float* C = new float[N * N];

    //매트릭스 초기화
    for (int i = 0; i < N * N; ++i) {
        A[i] = 1.0f;
        B[i] = 2.0f;
    }

    // 디바이스 메모리 할당
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, N * N * sizeof(float));
    hipMalloc(&d_B, N * N * sizeof(float));
    hipMalloc(&d_C, N * N * sizeof(float));

    // 호스트에서 디바이스로 데이터 복사
    hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // 定义 CUDA 核函数执行配置
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    auto start = std::chrono::steady_clock::now();

    do {
        matrixMultiplication<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);
        hipDeviceSynchronize();
        auto end = std::chrono::steady_clock::now();
        std::chrono::duration<double> elapsed_seconds = end - start;
        if (elapsed_seconds.count() >= 60) {
            break;
        }
    } while (true);

    auto end = std::chrono::steady_clock::now();
    std::chrono::duration<double> elapsed_seconds = end - start;
    std::cout << "Elapsed time: " << elapsed_seconds.count() << "s" << std::endl;

    // 디바이스의 결과를 다시 호스트로 복사
    hipMemcpy(C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // 디바이스 메모리 해제
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    delete[] A;
    delete[] B;
    delete[] C;

    return 0;
}

